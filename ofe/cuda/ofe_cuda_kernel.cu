#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__device__ __forceinline__ float atomicMaxFloat(float* addr, float value) {
    float old;
    old = !signbit(value) ? __int_as_float(atomicMax((int*)addr, __float_as_int(value))) :
        __uint_as_float(atomicMin((unsigned int*)addr, __float_as_uint(value)));
    return old;
}


__global__ void octree_feature_extractor_cuda_kernel(
        const float* faces,
        const bool* mask,
        const float* depth_map,
        const int* batch_id,
        const int num_faces,
        const int image_height,
        const int image_width,
        float* octree_feature) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_faces) {
        return;
    }
    const int ih = image_height;
    const int iw = image_width;
    const int bn = batch_id[i / 12];
    const int fn = i;

    const float* face = &faces[i * 9];

    /* pi[0], pi[1], pi[2] = leftmost, middle, rightmost points */
    int pi[3];
    if (face[0] < face[3]) {
        if (face[6] < face[0]) pi[0] = 2; else pi[0] = 0;
        if (face[3] < face[6]) pi[2] = 2; else pi[2] = 1;
    } else {
        if (face[6] < face[3]) pi[0] = 2; else pi[0] = 1;
        if (face[0] < face[6]) pi[2] = 2; else pi[2] = 0;
    }
    for (int k = 0; k < 3; k++) {
      if (pi[0] != k && pi[2] != k) {
          pi[1] = k;
      }
    }

    /* p[num][xyz]: x, y is normalized from [-1, 1] to [0, ih or iw - 1]. */
    float p[3][3];
    for (int num = 0; num < 3; num++) {
        for (int dim = 0; dim < 3; dim++) {
            if (dim == 0) {
                p[num][dim] = 0.5 * (face[3 * pi[num] + dim] * iw + iw - 1);
            } else if (dim == 1) {
                p[num][dim] = 0.5 * (face[3 * pi[num] + dim] * ih + ih - 1);
            } else {
                p[num][dim] = face[3 * pi[num] + dim];
            }
        }
    }
    if (p[0][0] == p[2][0]) return; // line, not triangle

    /* compute face_inv */
    float face_inv[9] = {
        p[1][1] - p[2][1], p[2][0] - p[1][0], p[1][0] * p[2][1] - p[2][0] * p[1][1],
        p[2][1] - p[0][1], p[0][0] - p[2][0], p[2][0] * p[0][1] - p[0][0] * p[2][1],
        p[0][1] - p[1][1], p[1][0] - p[0][0], p[0][0] * p[1][1] - p[1][0] * p[0][1]};

    float face_inv_denominator = (
        p[2][0] * (p[0][1] - p[1][1]) +
        p[0][0] * (p[1][1] - p[2][1]) +
        p[1][0] * (p[2][1] - p[0][1]));

    for (int k = 0; k < 9; k++) {
        face_inv[k] /= face_inv_denominator;
    }

    const int xi_min = max(ceil(p[0][0]), 0.);
    const int xi_max = min(p[2][0], iw - 1.0);

    for (int xi = xi_min; xi <= xi_max; xi++) {
        /* compute yi_min and yi_max */
        float yi1, yi2;
        if (xi <= p[1][0]) {
            if (p[1][0] - p[0][0] != 0) {
                yi1 = (p[1][1] - p[0][1]) / (p[1][0] - p[0][0]) * (xi - p[0][0]) + p[0][1];
            } else {
                yi1 = p[1][1];
            }
        } else {
            if (p[2][0] - p[1][0] != 0) {
                yi1 = (p[2][1] - p[1][1]) / (p[2][0] - p[1][0]) * (xi - p[1][0]) + p[1][1];
            } else {
                yi1 = p[1][1];
            }
        }
        yi2 = (p[2][1] - p[0][1]) / (p[2][0] - p[0][0]) * (xi - p[0][0]) + p[0][1];

        const int yi_min = max(0., ceil(min(yi1, yi2)));
        const int yi_max = min(max(yi1, yi2), ih - 1.0);

        for (int yi = yi_min; yi <= yi_max; yi++) {
            /* index in output buffers */
            int index = bn * ih * iw + yi * iw + xi;
            /* compute w = face_inv * p */
            float w[3];
            for (int k = 0; k < 3; k++) {
                w[k] = face_inv[3 * k + 0] * xi + face_inv[3 * k + 1] * yi + face_inv[3 * k + 2];
            }
            /* sum(w) -> 1, 0 < w < 1 */
            float w_sum = 0;
            for (int k = 0; k < 3; k++) {
                w[k] = min(max(w[k], 0.0), 1.0);
                w_sum += w[k];
            }
            for (int k = 0; k < 3; k++) w[k] /= w_sum;
            /* compute 1 / zp = sum(w / z) */
            const float zp = 1.0 / (w[0] / p[0][2] + w[1] / p[1][2] + w[2] / p[2][2]);
            const float zp_diff = zp - depth_map[index];
            const float float_mask = static_cast<float>(mask[index]);
            atomicMaxFloat(&octree_feature[(i / 12) * 2], zp_diff);
            atomicMaxFloat(&octree_feature[(i / 12) * 2 + 1], float_mask);
        }
    }
}


at::Tensor run_cuda(
        const at::Tensor& faces,
        const at::Tensor& mask,
        const at::Tensor& depth_map,
        const at::Tensor& batch_id,
        const int image_height,
        const int image_width) {

    const int num_faces = faces.size(0);
    const int threads = 512;

    auto float_opts = faces.options().dtype(at::kFloat);

    at::Tensor octree_feature = at::full({num_faces / 12, 2}, 0.0, float_opts);

    const dim3 blocks1 ((num_faces - 1) / threads +1);

    octree_feature_extractor_cuda_kernel<<<blocks1, threads>>>(
        faces.data_ptr<float>(),
        mask.data_ptr<bool>(),
        depth_map.data_ptr<float>(),
        batch_id.data_ptr<int>(),
        num_faces,
        image_height,
        image_width,
        octree_feature.data_ptr<float>());

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)  {
        printf("Error in forward_face_index_map: %s\n", hipGetErrorString(err));
    }

    return octree_feature;
}
